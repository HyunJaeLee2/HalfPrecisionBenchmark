#include "hip/hip_runtime.h"
#include "half2.h"

__global__ void MatMulKernel_half2(Matrix_half2 A, Matrix_half2 B, Matrix_half2 C) {
    __half2 Cvalue = __floats2half2_rn(0.0, 0.0);
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= A.height || col >= B.width_len) return;

    for (int e = 0; e < A.width_len; ++e)
        Cvalue = __hadd2( Cvalue, 
                        __hmul2( (A.half2_elements[row * A.width_len + e]), 
                                 (B.half2_elements[e * B.width_len + col]) ) 
                         );

    C.half2_elements[row * C.width_len + col] = Cvalue;
}


__global__ void float2DupHalf2_rn_kernel(int size, const float *buffIn, __half2 *buffOut)
{
    const int idx = (BLOCK_SIZE*blockIdx.x+threadIdx.x);
    if (idx >= size/sizeof(__half2)) return;
	
    __half2 val;
    val = __floats2half2_rn(float(buffIn[idx]), float(buffIn[idx]));
    buffOut[idx] = val;
}

__global__ void float2half2_rn_kernel(int size, const float *buffIn, __half2 *buffOut)
{
    const int idx = (BLOCK_SIZE *blockIdx.x / 2 + threadIdx.x) * 2; // divide by 2 because dimBlock is BlockSIZE / 2
    if (idx >= size/sizeof(__half2)) return;
	
    __half2 val;
    val = __floats2half2_rn(float(buffIn[idx]), float(buffIn[idx+1]));
    buffOut[idx / 2] = val;
}

void gpu_float2DupHalf2_rn(int size, const float *buffIn, __half2 *buffOut)
{
    int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    float2DupHalf2_rn_kernel<<<grid_size, BLOCK_SIZE>>> (size, buffIn, buffOut);
    hipDeviceSynchronize();
}


//size : length of buffIn
void gpu_float2half2_rn(int size, const float *buffIn, __half2 *buffOut)
{
    int dimBlock = BLOCK_SIZE / 2;
    int grid_size = (size + dimBlock - 1) / dimBlock;
    float2half2_rn_kernel<<<grid_size, dimBlock>>> (size, buffIn, buffOut);
    hipDeviceSynchronize();
}

__global__ void half22float_kernel(int size, const __half2 *buffIn, float *buffOut)
{
    const int idx = BLOCK_SIZE*blockIdx.x+threadIdx.x;
    if (idx >= size / sizeof(__half2)) return;
   
    float hi_float;
    float lo_float;
    asm("{.reg .f16 low,high;\n"
        "  mov.b32 {low,high},%1;\n"
        "  cvt.f32.f16 %0, low;}\n" : "=f"(lo_float) : "r"(buffIn[idx].x));

    asm("{.reg .f16 low,high;\n"
        "  mov.b32 {low,high},%1;\n"
        "  cvt.f32.f16 %0, high;}\n" : "=f"(hi_float) : "r"(buffIn[idx].x));
    buffOut[idx * 2] = lo_float;
    buffOut[idx * 2 + 1] = hi_float;
}

//size : length of buffIn
void gpu_half22float(int size, const __half2 *buffIn, float *buffOut)
{
    int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    half22float_kernel<<<grid_size, BLOCK_SIZE>>> (size, buffIn, buffOut);
    hipDeviceSynchronize();
}

void MatMul_half2(const Matrix A, const Matrix B, Matrix C) {
    //allocate temp mem to copy float
    Matrix d_tempA;
    d_tempA.width = A.width;
    d_tempA.height = A.height;
    int size_temp = A.width * A.height * sizeof(float);

    hipError_t err = hipMalloc(&d_tempA.elements, size_temp);
    //printf("CUDA malloc tempA: %s\n",hipGetErrorString(err));

    err = hipMemcpy(d_tempA.elements, A.elements, size_temp, hipMemcpyHostToDevice);
    //printf("Copy A to device: %s\n",hipGetErrorString(err));
   
    //allocate half2 mem
    Matrix_half2 d_A;
    d_A.width = A.width;
    d_A.width_len = A.width;
    d_A.height = A.height;
    int size = d_A.width_len * d_A.height * sizeof(__half2);
    
    err = hipMalloc(&d_A.half2_elements, size);
    //printf("CUDA malloc A: %s (%d)\n",hipGetErrorString(err), size);
    
    gpu_float2DupHalf2_rn(size, d_tempA.elements, d_A.half2_elements);
    //gpu_float2half2_rn(size, d_tempA.elements, d_A.half2_elements);

    //allocate temp mem to copy float
    Matrix d_tempB;
    d_tempB.width = B.width;
    d_tempB.height = B.height;
    size_temp = B.width * B.height * sizeof(float);

    err = hipMalloc(&d_tempB.elements, size_temp);
    //printf("CUDA malloc tempB: %s (%d)\n",hipGetErrorString(err), size_temp);

    err = hipMemcpy(d_tempB.elements, B.elements, size_temp, hipMemcpyHostToDevice);
    //printf("Copy B to device: %s\n",hipGetErrorString(err));
    
    //allocate half2 mem
    Matrix_half2 d_B;
    d_B.width = B.width;
    d_B.width_len = B.width / 2;
    d_B.height = B.height;
    size = d_B.width_len * d_B.height * sizeof(__half2);

    err = hipMalloc(&d_B.half2_elements, size);
    //printf("CUDA malloc B: %s (%d)\n",hipGetErrorString(err), size);

    gpu_float2half2_rn(size * 2, d_tempB.elements, d_B.half2_elements);
    
    // Allocate C in device memory
    Matrix_half2 d_C;
    d_C.width = C.width;
    d_C.width_len = C.width / 2;
    d_C.height = C.height;
    size = d_C.width_len * d_C.height * sizeof(__half2);

    err = hipMalloc(&d_C.half2_elements, size);
    //printf("CUDA malloc C: %s\n",hipGetErrorString(err));

    hipFree(d_tempA.elements);
    hipFree(d_tempB.elements);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE / 2, BLOCK_SIZE);
    dim3 dimGrid((d_B.width_len + dimBlock.x - 1) / dimBlock.x,
            (A.height + dimBlock.y - 1) / dimBlock.y);

    long t = timer_get();
    MatMulKernel_half2<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    err = hipDeviceSynchronize();
    fprintf(stderr,   "[half2  ]\t%9ld\n", timer_get() - t);
    
    //printf("Run kernel: %s\n", hipGetErrorString(err));
    
    //allocate temp mem to copy half2
    Matrix d_tempC;
    d_tempC.width = C.width;
    d_tempC.height = C.height;
    size_temp = C.width * C.height * sizeof(float); 

    err = hipMalloc(&d_tempC.elements, size_temp);
    //printf("CUDA malloc tempC: %s\n",hipGetErrorString(err));

    // Read C from device memory
    gpu_half22float(size, d_C.half2_elements, d_tempC.elements);

    err = hipMemcpy(C.elements, d_tempC.elements, size_temp, hipMemcpyDeviceToHost);
    //printf("Copy C off of device: %s\n",hipGetErrorString(err));

    // Free device memory
    hipFree(d_A.half2_elements);
    hipFree(d_B.half2_elements);
    hipFree(d_C.half2_elements);
    hipFree(d_tempC.elements);
}
