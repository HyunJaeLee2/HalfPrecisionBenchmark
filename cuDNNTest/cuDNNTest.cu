#include "hip/hip_runtime.h"
#include "util.hpp"
#include "hip/hip_runtime.h"
#include "cuDNNTest.h"

using namespace std;

const int batch_count = 1;
hipdnnHandle_t cudnnHandle;
bool isHalfPrecision;

void free_layer(conv_layer *layer)
{
	checkCUDNN(hipdnnDestroyTensorDescriptor(layer->inTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(layer->outTensorDesc));

	checkCUDNN(hipdnnDestroyConvolutionDescriptor(layer->convDesc));
	checkCUDNN(hipdnnDestroyFilterDescriptor(layer->filterDesc));

    if(layer->d_inData) checkCUDA(hipFree(layer->d_inData));
    if(layer->d_outData) checkCUDA(hipFree(layer->d_outData));;
    if(layer->d_filterData) checkCUDA(hipFree(layer->d_filterData));

    if(layer->d_half_inData) checkCUDA(hipFree(layer->d_half_inData));
    if(layer->d_half_outData) checkCUDA(hipFree(layer->d_half_outData));;
    if(layer->d_half_filterData) checkCUDA(hipFree(layer->d_half_filterData));
    
    if(layer->workSpace) checkCUDA(hipFree(layer->workSpace));
}

conv_layer initFirstLayerWithRandom(char *argv[])
{
	float *inData, *filterData;
    int inSize, outSize, filterSize;
    
    conv_layer layer = {0};

    //set padding and stride to 1 as default
    layer.padding_h = layer.padding_w = layer.stride_vertical = layer.stride_horizontal = 1;

    layer.in_height = layer.in_width = atoi(argv[1]);
    layer.in_channel = atoi(argv[2]);
    layer.filter_height = layer.filter_width = atoi(argv[3]);
    layer.filter_num = atoi(argv[4]);
    
    //Init Size 
    inSize = batch_count * layer.in_channel * layer.in_height * layer.in_width;
    filterSize = layer.filter_num * layer.in_channel * layer.filter_height * layer.filter_width;
    outSize = batch_count * layer.filter_num * layer.in_height * layer.in_width;
    layer.outSize = outSize;
    layer.inSize = inSize;
    layer.filterSize = filterSize;

    //Init Data
    inData = (float *)malloc(sizeof(float) * inSize);
    filterData = (float *)malloc(sizeof(float) * filterSize);

    initWithRandom4D(inData, batch_count, layer.in_channel, layer.in_height, layer.in_width);
    //print4D("In Data", inData, batch_count, layer.in_channel, layer.in_height, layer.in_width);

    initWithRandom4D(filterData, layer.filter_num, layer.in_channel, layer.filter_height, layer.filter_width);
    //print4D("Filter Data", filterData, layer.filter_num, layer.in_channel, layer.filter_height, layer.filter_width);
    
    checkCUDA(hipMalloc((void**)&layer.d_inData, inSize * sizeof(float)));
	checkCUDA(hipMalloc((void**)&layer.d_filterData, filterSize * sizeof(float)));
    checkCUDA(hipMalloc((void**)&layer.d_outData, outSize * sizeof(float)));
    
	checkCUDA(hipMemcpy(layer.d_inData, inData, inSize * sizeof(float), hipMemcpyHostToDevice));
	checkCUDA(hipMemcpy(layer.d_filterData, filterData, filterSize * sizeof(float), hipMemcpyHostToDevice));

	checkCUDNN(hipdnnCreateTensorDescriptor(&layer.inTensorDesc));
	checkCUDNN(hipdnnCreateTensorDescriptor(&layer.outTensorDesc));
	checkCUDNN(hipdnnCreateFilterDescriptor(&layer.filterDesc));
	checkCUDNN(hipdnnCreateConvolutionDescriptor(&layer.convDesc));
    
    const int convDims = 2;
    int padA[convDims] = {layer.padding_h, layer.padding_w};
    int filterStrideA[convDims] = {layer.stride_vertical, layer.stride_horizontal};
    int upscaleA[convDims] = {1, 1};
    int out_n, out_c, out_h, out_w;
    
    if(!isHalfPrecision)
    {
        checkCUDNN(hipdnnSetTensor4dDescriptor(layer.inTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_count, layer.in_channel, layer.in_height, layer.in_width));
        checkCUDNN(hipdnnSetFilter4dDescriptor(layer.filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, layer.filter_num, layer.in_channel, layer.filter_height, layer.filter_width));
        
        hipdnnSetConvolutionNdDescriptor(layer.convDesc, convDims, padA, filterStrideA, upscaleA, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT); 
        
        checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(layer.convDesc, layer.inTensorDesc, layer.filterDesc, &out_n, &out_c, &out_h, &out_w));
        
        checkCUDNN(hipdnnSetTensor4dDescriptor(layer.outTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));
    }
    else
    {
        checkCUDA(hipMalloc((void**)&layer.d_half_inData, inSize * sizeof(__half)));
        checkCUDA(hipMalloc((void**)&layer.d_half_filterData, filterSize * sizeof(__half)));
        checkCUDA(hipMalloc((void**)&layer.d_half_outData, outSize * sizeof(__half)));

        gpu_float2half_rn(inSize, layer.d_inData, layer.d_half_inData);
        gpu_float2half_rn(filterSize, layer.d_filterData, layer.d_half_filterData);

        checkCUDNN(hipdnnSetTensor4dDescriptor(layer.inTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_HALF, batch_count, layer.in_channel, layer.in_height, layer.in_width));
        checkCUDNN(hipdnnSetFilter4dDescriptor(layer.filterDesc, HIPDNN_DATA_HALF, HIPDNN_TENSOR_NCHW, layer.filter_num, layer.in_channel, layer.filter_height, layer.filter_width));
        
        hipdnnSetConvolutionNdDescriptor(layer.convDesc, convDims, padA, filterStrideA, upscaleA, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_HALF); 
        
        checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(layer.convDesc, layer.inTensorDesc, layer.filterDesc, &out_n, &out_c, &out_h, &out_w));
        
        checkCUDNN(hipdnnSetTensor4dDescriptor(layer.outTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_HALF, out_n, out_c, out_h, out_w));
    }
    printf("conv out shape (n x c x h x w) = (%d x %d x %d x %d)\n", out_n, out_c, out_h, out_w);

	checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
				layer.inTensorDesc,
				layer.filterDesc,
				layer.convDesc,
				layer.outTensorDesc,
				HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
				0,
				&layer.algo
				));

	cout << "Fastest layer.algorithm for conv0 = " << layer.algo << endl;

	checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
				layer.inTensorDesc,
				layer.filterDesc,
				layer.convDesc,
				layer.outTensorDesc,
				layer.algo,
				&layer.sizeInBytes));

	cout << "sizeInBytes " << layer.sizeInBytes << endl;

	if (layer.sizeInBytes != 0) checkCUDA(hipMalloc(&layer.workSpace, layer.sizeInBytes));

    free(inData);
    free(filterData);
    
    return layer;
}

int main(int argc, char* argv[])
{
    long time_accum = 0;
    float *outData;
	conv_layer conv1;

    if(argc < 6)
    {
        fprintf(stderr, "usage : ./test [Input Height/Width] [Channel] [Filter Height/Width] [Filter Num] [float/half]\n");
        return 0;
    }
    
    if(strcmp(argv[5], "half") == 0) 
        isHalfPrecision = true;
    else 
        isHalfPrecision = false;
    
    
    checkCUDNN(hipdnnCreate(&cudnnHandle));
    
    conv1 = initFirstLayerWithRandom(argv);   
	
    outData = (float *)malloc(sizeof(float) * conv1.outSize);

	float alpha = 1.0f;
	float beta = 0.0f;
   
    long t = timer_get();
    if(!isHalfPrecision)
    {
        checkCUDNN(hipdnnConvolutionForward(cudnnHandle,
                    &alpha,
                    conv1.inTensorDesc,
                    conv1.d_inData,
                    conv1.filterDesc,
                    conv1.d_filterData,
                    conv1.convDesc,
                    conv1.algo,
                    conv1.workSpace,
                    conv1.sizeInBytes,
                    &beta,
                    conv1.outTensorDesc,
                    conv1.d_outData));
        
        checkCUDA(hipDeviceSynchronize());
        time_accum += (timer_get() - t);

        checkCUDA(hipMemcpy(outData, conv1.d_outData, sizeof(float)* conv1.outSize, hipMemcpyDeviceToHost));
		//print("conv out", outData, conv1.filter_num, conv1.in_height, conv1.in_width);
        print4D("conv out", outData, 1, conv1.filter_num, conv1.in_height, conv1.in_width);

        fprintf(stderr,   "[float  ]\t%9ld\n", time_accum);
    }
    else
    {
        if(conv1.d_inData) {checkCUDA(hipFree(conv1.d_inData)); conv1.d_inData = NULL;}
        if(conv1.d_outData){checkCUDA(hipFree(conv1.d_outData)); conv1.d_outData = NULL;}
        if(conv1.d_filterData) {checkCUDA(hipFree(conv1.d_filterData)); conv1.d_filterData = NULL;};
        
        checkCUDNN(hipdnnConvolutionForward(cudnnHandle,
                    &alpha,
                    conv1.inTensorDesc,
                    conv1.d_half_inData,
                    conv1.filterDesc,
                    conv1.d_half_filterData,
                    conv1.convDesc,
                    conv1.algo,
                    conv1.workSpace,
                    conv1.sizeInBytes,
                    &beta,
                    conv1.outTensorDesc,
                    conv1.d_half_outData));

        checkCUDA(hipDeviceSynchronize());
        time_accum += (timer_get() - t);

        //d_outData was freed then reallocated to optimize memory
        checkCUDA(hipMalloc((void**)&conv1.d_outData, conv1.outSize * sizeof(float)));
        
        gpu_half2float(conv1.outSize, conv1.d_half_outData, conv1.d_outData);
		checkCUDA(hipMemcpy(outData, conv1.d_outData, sizeof(float)* conv1.outSize, hipMemcpyDeviceToHost));
		print4D("conv out", outData, 1, conv1.filter_num, conv1.in_height, conv1.in_width);

		fprintf(stderr,   "[half  ]\t%9ld\n", time_accum);
    }

    free_layer(&conv1);

	checkCUDA(hipDeviceSynchronize());
	return 0;
}
