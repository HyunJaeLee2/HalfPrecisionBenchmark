#include <iostream>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

using namespace std;

void checkCUDNN(hipdnnStatus_t status)
{
 if (status != HIPDNN_STATUS_SUCCESS)
  cout << "[ERROR] CUDNN " << status << endl;
}

void checkCUDA(hipError_t error)
{
 if (error != hipSuccess)
  cout << "[ERROR] CUDA " << error << endl;
}

void print(char* title, float* src, int filter_num, int h, int w)
{
 cout << title << endl;
 for (int i = 0; i < filter_num; i++) {
  for (int y = 0; y < h; y++) {
   for (int x = 0; x < w; x++) {
    printf("%.0f ", src[i*h * w + y * w + x]);
   }
   cout << endl;
  }
  cout << endl;
 }
}

int main()
{
 const int batch_count = 1;//입력 데이터 갯수, 배치사이즈
 const int in_channel = 2;//입력 데이터의 채널 수
 const int in_height = 4;//입력 데이터의 세로 길이
 const int in_width = 4;//입력 데이터의 가로 길이
 const int out_channel = 2;//출력 클래스 수
 const int filter_width = 3;//컨볼루션 필터(가중치)의 가로 길이
 const int filter_height = 3;//컨볼루션 필터(가중치)의 세로 길이
 const int filter_num = 1;//컨볼루션 필터(가중치) 갯수
 const int padding_w = 1;//컨볼루션 패딩. 필터의 가로 세로 길이가 3이고 패딩이 1,1 이면 SAME Convolution이 된다
 const int padding_h = 1;
 const int stride_horizontal = 1;//컨볼루션 스트라이드
 const int stride_vertical = 1;
 const int pool_window_w = 2;
 const int pool_window_h = 2;
 const int pool_stride_horizontal = 2;
 const int pool_stride_vertical = 2;
 const int pool_padding_horizontal = 0;
 const int pool_padding_vertical = 0;
 const int pool_w = in_width / pool_stride_horizontal;
 const int pool_h = in_height / pool_stride_vertical;
 const int src_len = batch_count*filter_num*in_height*in_width;
 const int pool_len = batch_count*filter_num*pool_w * pool_h;
 float inData_NCHW[batch_count][in_channel][in_height][in_width];//host 입력 데이터 NCHW
 float inData_NHWC[batch_count][in_height][in_width][in_channel];//host 입력 데이터 NHWC
 float outData[batch_count][filter_num][in_height][in_width];//host 출력 데이터
 float *hostArray = new float[src_len];
 float *inData_d;//device 입력 데이터
 float *outData_d, *outData1_d;//device 출력 데이터
 float *filterData_d;//device 컨볼루션 필터 데이터
 float *filterData2_d;//device FCN 필터 데이터
 float *biasData_d;
 void* workSpace;//CUDNN이 작업 중에 사용할 버퍼 메모리

 //입력 데이터 셋팅
 for (int i = 0; i < in_channel; i++) {
  for (int y = 0; y < in_height; y++) {
   for (int x = 0; x < in_width; x++) {
    inData_NCHW[0][i][y][x] = i * in_channel * in_height * in_width + y * in_width + x;
   }
  }
 }
 //입력 데이터 형태 변환
 for (int i = 0; i < in_channel; i++) {
  for (int y = 0; y < in_height; y++) {
   for (int x = 0; x < in_width; x++) {
    inData_NHWC[0][y][x][i] = inData_NCHW[0][i][y][x];
   }
  }
 }

 //필터(가중치) 셋팅
 float filterData[filter_num][in_channel][filter_height][filter_width] = {
  {
   { { 0.0f, 0.0f, 0.0f }, { 0.0f, 1.0f, 0.0f }, { 0.0f, 0.0f, 0.0f } },
   { { 0.0f, 0.0f, 0.0f }, { 0.0f, 0.0f, 1.0f }, { 0.0f, 0.0f, 0.0f } }
  }
 };

 //Fully connected Layer 가중치
float filterData2[out_channel][filter_num][pool_h][pool_w] = {
 {{ { 0.1f, 0.1f }, { 0.1f, 0.1f } }},
 {{ { 0.2f, 0.2f }, { 0.2f, 0.2f } }}
};

 float biasData[filter_num] = {
  -20
 };

 cout << "in_NCHW" << endl;
 for (int i = 0; i < in_channel; i++) {
  for (int y = 0; y < in_height; y++) {
   for (int x = 0; x < in_width; x++) {
    printf("%.0f ", inData_NCHW[0][i][y][x]);
   }
   cout << endl;
  }
  cout << endl;
 }

 cout << "in_NHWC" << endl;
 for (int y = 0; y < in_height; y++) {
  for (int x = 0; x < in_width; x++) {
   for (int i = 0; i < in_channel; i++) {
    printf("%.0f ", inData_NHWC[0][y][x][i]);
   }
   cout << endl;
  }
  cout << endl;
 }

 cout << "weights" << endl;
 for (int n = 0; n < filter_num; n++)
 {
  for (int i = 0; i < in_channel; i++) {
   for (int y = 0; y < filter_height; y++) {
    for (int x = 0; x < filter_width; x++) {
     printf("%.1f ", filterData[n][i][y][x]);
    }
    cout << endl;
   }
   cout << endl;
  }
 }

 //GPU 메모리 할당
 checkCUDA(hipMalloc((void**)&inData_d, sizeof(inData_NCHW)));
 checkCUDA(hipMalloc((void**)&outData_d, sizeof(outData)));
 checkCUDA(hipMalloc((void**)&outData1_d, sizeof(outData)));
 checkCUDA(hipMalloc((void**)&filterData_d, sizeof(filterData)));
 checkCUDA(hipMalloc((void**)&biasData_d, sizeof(biasData)));
 checkCUDA(hipMalloc((void**)&filterData2_d, sizeof(filterData2)));

 //CPU 데이터를 GPU 메모리로 복사
 //NHWC 와 NCHW 중에 선택합니다.
 //checkCUDA(hipMemcpy(inData_d, inData_NCHW, sizeof(inData_NCHW), hipMemcpyHostToDevice));
 checkCUDA(hipMemcpy(inData_d, inData_NHWC, sizeof(inData_NHWC), hipMemcpyHostToDevice));
 checkCUDA(hipMemcpy(filterData_d, filterData, sizeof(filterData), hipMemcpyHostToDevice));
 checkCUDA(hipMemcpy(filterData2_d, filterData2, sizeof(filterData2), hipMemcpyHostToDevice));
 checkCUDA(hipMemcpy(biasData_d, biasData, sizeof(biasData), hipMemcpyHostToDevice));

 //CUDNN 배열
 hipdnnHandle_t cudnnHandle;// CUDNN을 사용하기 위한 핸들러
 hipdnnTensorDescriptor_t inTensorDesc, outTensorDesc, biasTensorDesc, poolOutTensorDesc, sftTensorDesc;//데이터 구조체 선언
 hipdnnFilterDescriptor_t filterDesc, filterDesc2;//필터 구조체 선언
 hipdnnConvolutionDescriptor_t convDesc;//컨볼루션 구조체 선언 
 hipdnnConvolutionDescriptor_t convDesc2;//컨볼루션 구조체 선언 
 hipdnnPoolingDescriptor_t poolDesc;//풀링 구조체 선언
 hipdnnActivationDescriptor_t actDesc;//활성함수 구조체 선언

 //할당
 checkCUDNN(hipdnnCreate(&cudnnHandle));
 checkCUDNN(hipdnnCreateTensorDescriptor(&inTensorDesc));
 checkCUDNN(hipdnnCreateTensorDescriptor(&outTensorDesc));
 checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensorDesc));
 checkCUDNN(hipdnnCreateTensorDescriptor(&poolOutTensorDesc));
 checkCUDNN(hipdnnCreateTensorDescriptor(&sftTensorDesc));
 checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
 checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc2));
 checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
 checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc2));
 checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
 checkCUDNN(hipdnnCreateActivationDescriptor(&actDesc));


 //초기화
 //inData_NCHW 정보 - 구조가 [Number][Channel][Height][Width] 형태임을 알려줌
 //checkCUDNN(hipdnnSetTensor4dDescriptor(inTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_count, in_channel, in_height, in_width));
 //inData_NHWC 정보 - 구조가 [Number][Height][Width][Channel] 형태임을 알려줌
 checkCUDNN(hipdnnSetTensor4dDescriptor(inTensorDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, batch_count, in_channel, in_height, in_width));
 checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, filter_num, in_channel, filter_height, filter_width));
 checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc2, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, out_channel, filter_num, pool_h, pool_w));
 //컨볼루션의 패딩, 스트라이드, 컨볼루션 모드 등을 셋팅
 checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padding_h, padding_w, stride_vertical, stride_horizontal, 1, 1, HIPDNN_CROSS_CORRELATION));
 //풀리 커넥티드 네트워크 셋업
 checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc2, 0, 0, 2, 2, 1, 1, HIPDNN_CROSS_CORRELATION));
 //바이어스 셋업
 checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, filter_num, 1, 1));
 //풀링 셋업 
 checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, pool_window_h, pool_window_w, pool_padding_vertical, pool_padding_horizontal, pool_stride_vertical, pool_stride_horizontal));
 //활성함수 Relu 셋업
 checkCUDNN(hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0));

 int out_n, out_c, out_h, out_w;
 //입력데이터를 위에서 셋팅한 대로 컨볼루션 했을때 출력 데이터의 구조 알아내기
 checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, inTensorDesc, filterDesc, &out_n, &out_c, &out_h, &out_w));
 printf("conv out shape (n x c x h x w) = (%d x %d x %d x %d)\n", out_n, out_c, out_h, out_w);
 checkCUDNN(hipdnnSetTensor4dDescriptor(outTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));

 //풀링 결과 구조 확인
 checkCUDNN(hipdnnGetPooling2dForwardOutputDim(poolDesc, outTensorDesc, &out_n, &out_c, &out_h, &out_w));
 printf("pool out shape (n x c x h x w) = (%d x %d x %d x %d)\n", out_n, out_c, out_h, out_w);
 //풀링 결과 구조 셋업
 checkCUDNN(hipdnnSetTensor4dDescriptor(poolOutTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));

 //FCN 결과 구조 확인
 checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc2, poolOutTensorDesc, filterDesc2, &out_n, &out_c, &out_h, &out_w));
 printf("conv2 out shape (n x c x h x w) = (%d x %d x %d x %d)\n", out_n, out_c, out_h, out_w);
 checkCUDNN(hipdnnSetTensor4dDescriptor(sftTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w));

 //입력과 필터, 컨볼루션 패딩, 스트라이드가 위와 같이 주어졌을때 가장 빠른 알고리즘이 무엇인지를 알아내기
 hipdnnConvolutionFwdAlgo_t algo;
 checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
  inTensorDesc,
  filterDesc,
  convDesc,
  outTensorDesc,
  HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
  0,
  &algo
  ));

 cout << "Fastest algorithm for conv0 = " << algo << endl;

 hipdnnConvolutionFwdAlgo_t algo2;
 checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnnHandle,
  poolOutTensorDesc,
  filterDesc2,
  convDesc2,
  sftTensorDesc,
  HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
  0,
  &algo2
  ));

 cout << "Fastest algorithm for conv1 = " << algo2 << endl;

 //위에서 알아낸 가장 빠른 알고리즘을 사용할 경우 계산과정에서 필요한 버퍼 데이터의 크기를 알아내기
 size_t sizeInBytes = 0;
 checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
  inTensorDesc,
  filterDesc,
  convDesc,
  outTensorDesc,
  algo,
  &sizeInBytes));

 cout << "sizeInBytes " << sizeInBytes << endl;
 //계산과정에서 버퍼 데이터가 필요한 경우가 있다면 메모리 할당
 if (sizeInBytes != 0) checkCUDA(hipMalloc(&workSpace, sizeInBytes));

 float alpha = 1.0f;
 float beta = 0.0f;
 //컨볼루션 시작
 //alpha와 beta는 "output = alpha * Op(inputs) + beta * output" 에 사용됨
 //일반 컨볼루션은 output =   1   *    inputs 
 //그래서          output =   1   * Op(inputs) +   0  * output 이 되도록 alpha와 beta를 1,0으로 셋팅함
 checkCUDNN(hipdnnConvolutionForward(cudnnHandle,
  &alpha,
  inTensorDesc,
  inData_d,
  filterDesc,
  filterData_d,
  convDesc,
  algo,
  workSpace,
  sizeInBytes,
  &beta,
  outTensorDesc,
  outData_d));

 checkCUDA(hipMemcpy(hostArray, outData_d, sizeof(float)* src_len, hipMemcpyDeviceToHost));
 print("conv out", hostArray, filter_num, in_height, in_width);

 //Add Bias 
 beta = 1.0f;
 checkCUDNN(hipdnnAddTensor(cudnnHandle, &alpha, biasTensorDesc, biasData_d, &beta, outTensorDesc, outData_d));
 checkCUDA(hipMemcpy(hostArray, outData_d, sizeof(float)* src_len, hipMemcpyDeviceToHost));
 print("Add bias out", hostArray, filter_num, in_height, in_width);

 //Activation - Relu
 beta = 0.0f;
 checkCUDNN(hipdnnActivationForward(cudnnHandle, actDesc, &alpha, outTensorDesc, outData_d, &beta, outTensorDesc, outData1_d));
 checkCUDA(hipMemcpy(hostArray, outData1_d, sizeof(float)* src_len, hipMemcpyDeviceToHost));
 print("Activation - Relu out", hostArray, filter_num, in_height, in_width);

 //Pooling
 checkCUDNN(hipdnnPoolingForward(cudnnHandle, poolDesc, &alpha, outTensorDesc, outData1_d, &beta, poolOutTensorDesc, outData_d));
 checkCUDA(hipMemcpy(hostArray, outData_d, sizeof(float)* pool_len, hipMemcpyDeviceToHost));
 print("pool out", hostArray, filter_num, pool_h, pool_w);

 //FC
 checkCUDNN(hipdnnConvolutionForward(cudnnHandle,
  &alpha,
  poolOutTensorDesc,
  outData_d,
  filterDesc2,
  filterData2_d,
  convDesc2,
  algo2,
  workSpace,
  sizeInBytes,
  &beta,
  sftTensorDesc,
  outData1_d));

 checkCUDA(hipMemcpy(hostArray, outData1_d, sizeof(float)* out_channel, hipMemcpyDeviceToHost));
 print("FCN out", hostArray, out_channel, 1, 1);

 //Softmax
 hipdnnSoftmaxForward(cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &alpha, sftTensorDesc, outData1_d, &beta, sftTensorDesc, outData_d);
 checkCUDA(hipMemcpy(hostArray, outData_d, sizeof(float)* out_channel, hipMemcpyDeviceToHost));
 print("Softmax out", hostArray, out_channel, 1, 1);

 //메모리 해제
 checkCUDNN(hipdnnDestroyTensorDescriptor(inTensorDesc));
 checkCUDNN(hipdnnDestroyTensorDescriptor(outTensorDesc));
 checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensorDesc));
 checkCUDNN(hipdnnDestroyTensorDescriptor(poolOutTensorDesc));
 checkCUDNN(hipdnnDestroyTensorDescriptor(sftTensorDesc));

 checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
 checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc2));
 checkCUDNN(hipdnnDestroyPoolingDescriptor(poolDesc));
 checkCUDNN(hipdnnDestroyFilterDescriptor(filterDesc));
 checkCUDNN(hipdnnDestroyFilterDescriptor(filterDesc2));
 checkCUDNN(hipdnnDestroyActivationDescriptor(actDesc));
 checkCUDNN(hipdnnDestroy(cudnnHandle));

 checkCUDA(hipFree(inData_d));
 checkCUDA(hipFree(outData_d));;
 checkCUDA(hipFree(filterData_d));
 checkCUDA(hipDeviceSynchronize());
 return 0;
}
